#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "cuda_utils.h"
#include "timer.c"

typedef float dtype;


__global__ 
void matTrans(dtype* AT, dtype* A, int N)  
{
	/* Fill your code here */
	int tile_dim = 32;
	int block_row = 8;
	__shared__  dtype tile[TILE_SIZE];

	int x = blockIdx.x * tile_dim + threadIdx.x;
	int y = blockIdx.y * tile_dim + threadIdx.y;
	int width = gridDim.x * tile_dim;

	//split into 32*32 tiles
	for(int i = 0; i < tile_dim; i+= block_row){
		tile[(threadIdx.y + i) * tile_dim + threadIdx.x] = A[(y+i) * width + x];
	}

	__syncthreads ();

}

void
parseArg (int argc, char** argv, int* N)
{
	if(argc == 2) {
		*N = atoi (argv[1]);
		assert (*N > 0);
	} else {
		fprintf (stderr, "usage: %s <N>\n", argv[0]);
		exit (EXIT_FAILURE);
	}
}


void
initArr (dtype* in, int N)
{
	int i;

	for(i = 0; i < N; i++) {
		in[i] = (dtype) rand () / RAND_MAX;
	}
}

void
cpuTranspose (dtype* A, dtype* AT, int N)
{
	int i, j;

	for(i = 0; i < N; i++) {
		for(j = 0; j < N; j++) {
			AT[j * N + i] = A[i * N + j];
		}
	}
}

int
cmpArr (dtype* a, dtype* b, int N)
{
	int cnt, i;

	cnt = 0;
	for(i = 0; i < N; i++) {
		if(abs(a[i] - b[i]) > 1e-6) cnt++;
	}

	return cnt;
}



void
gpuTranspose (dtype* A, dtype* AT, int N)
{
	struct stopwatch_t* timer = NULL;
	long double t_gpu;
	dtype *i_data, *o_data;		//input data and outdata
	int TILE_DIM = 32;
	//defining the block and number of threads
	dim3 gb(N/TILE_DIM, N/TILE_DIM, 1);
	dim3 tb(TILE_DIM, 8, 1);

	//Allocating the memory for the input and output matrix
	CUDA_CHECK_ERROR(hipMalloc(&i_data, N*N*sizeof(dtype)));
	CUDA_CHECK_ERROR(hipMalloc(&o_data, N*N*sizeof(dtype)));
	CUDA_CHECK_ERROR(hipMemcpy(i_data, A, N*N*sizeof(dtype), hipMemcpyHostToDevice));

	/* Setup timers */
	stopwatch_init ();
	timer = stopwatch_create ();

	stopwatch_start (timer);

	/* run your kernel here */
	matTrans <<<gb, tb>>> (o_data, i_data, N);

	hipDeviceSynchronize ();
	t_gpu = stopwatch_stop (timer);
	fprintf (stderr, "GPU transpose: %Lg secs ==> %Lg billion elements/second\n",
	t_gpu, (N * N) / t_gpu * 1e-9 );

	CUDA_CHECK_ERROR(hipMemcpy (AT, o_data, N*N*sizeof(dtype),hipMemcpyDeviceToHost));

}

int 
main(int argc, char** argv)
{
  /* variables */
	dtype *A, *ATgpu, *ATcpu;
  int err;

	int N;

  struct stopwatch_t* timer = NULL;
  long double t_cpu;


	N = -1;
	parseArg (argc, argv, &N);

  /* input and output matrices on host */
  /* output */
  ATcpu = (dtype*) malloc (N * N * sizeof (dtype));
  ATgpu = (dtype*) malloc (N * N * sizeof (dtype));

  /* input */
  A = (dtype*) malloc (N * N * sizeof (dtype));

	initArr (A, N * N);

	/* GPU transpose kernel */
	gpuTranspose (A, ATgpu, N);

  /* Setup timers */
  stopwatch_init ();
  timer = stopwatch_create ();

	stopwatch_start (timer);
  /* compute reference array */
	cpuTranspose (A, ATcpu, N);
  t_cpu = stopwatch_stop (timer);
  fprintf (stderr, "Time to execute CPU transpose kernel: %Lg secs\n",
           t_cpu);

  /* check correctness */
	err = cmpArr (ATgpu, ATcpu, N * N);
	if(err) {
		fprintf (stderr, "Transpose failed: %d\n", err);
	} else {
		fprintf (stderr, "Transpose successful\n");
	}

	free (A);
	free (ATgpu);
	free (ATcpu);

  return 0;
}
